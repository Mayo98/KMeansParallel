#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include "Cluster.h"
#include "Point.h"
#include <cmath>
#include <vector>
#include <string>
#include <fstream>
#include <chrono>
#include <filesystem>
#include "SequentialKMeans.h"
#include <algorithm>


#define N 400000
#define TPB 128
static std::vector<int> used_pointIds;
using namespace std;

__host__ void readPoints(float *h_xval, float *h_yval, int* h_clusters) {

    std::string tmp = "";
    string line;
    //std::cout << std::filesystem::current_path().string()  << std::endl;
    ifstream infile("../cmake-build-debug/input1.txt");
    if (!infile.is_open()) {
        cout << "Error: Failed to open file." << endl;
        return;
    }
    int j = 0;
    while (getline(infile, line)) {
        for (int i = 0; i < static_cast<int>(line.length()); i++) {
            if ((48 <= static_cast<int>(line[i]) && static_cast<int>(line[i]) <= 57) || line[i] == '.' ||
                line[i] == '+' || line[i] == '-' || line[i] == 'e') {
                tmp += line[i];

            } else if (!tmp.empty()) {


                h_xval[j] = std::stod(tmp);

                //xval.push_back(std::stod(tmp));
                tmp = "";
            }
        }
        if (!tmp.empty()) {

           h_yval[j] = std::stod(tmp);
            //yval.push_back(std::stod(tmp));
            tmp = "";
        }
       h_clusters[j] = -1; //inizializzo anche il vettore clusters a -1
       j++;
    }
    //std::cout << "dim clusters: " << h_clusters << std::endl;

    cout << "\nDataset fetched!" << endl
         << endl;
    //std::cout << "Punti totali letti : " << xval.size() << std::endl;
}



static bool first = true;
std::vector<int>indexGenerator(int K, int total_points){
    //mem punti gia usati per init cluster
    if (first) {
        for (int i = 0; i < K; i++) {
            while (true) {
                int index = rand() % total_points;

                //check se index gia usato per un altro cluster

                if (std::find(used_pointIds.begin(), used_pointIds.end(), index) ==
                    used_pointIds.end()) {
                    used_pointIds.push_back(index);
                    //creo un cluster con avete centroide il punto attuale
                    break;
                }
            }
        }
        first = false;
    }
    return used_pointIds;
}
__global__ void clusterAssignment(const float *d_xval, const float *d_yval, int *d_clusterval, const float *d_centroidX, const float *d_centroidY, int K, bool *d_done){
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    *d_done = true;
    if (idx >= N) return;
    float min_dist = INFINITY;
    int closest_centroid = 0;

    float dist;
    for(int i = 0; i < K; i++)
    {
        float sum = 0.0;
        sum += pow(d_centroidX[i] - d_xval[idx], 2.0);
        sum += pow(d_centroidY[i] - d_yval[idx], 2.0);
        dist = sqrt(sum);
        if(dist < min_dist){
            min_dist = dist;
            closest_centroid = i;
        }
    }
    //assegno id-cluster al thread corrente
    if( d_clusterval[idx] != closest_centroid) {
        d_clusterval[idx] = closest_centroid;
        *d_done = false;
    }
}

__global__ void clusterPointsSum(float* d_xval, float* d_yval, int* d_clusterVal, float* d_clusterSumX, float* d_clusterSumY, int* d_clusterSize){
    //indice del thread a livello grid

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    int clusterId = d_clusterVal[idx];
    //sommo tutti i punti appartenenti ai clusters
    atomicAdd(&(d_clusterSumX[clusterId]), d_xval[idx]);
    atomicAdd(&(d_clusterSumY[clusterId]), d_yval[idx]);
    atomicAdd(&(d_clusterSize[clusterId]), 1);

}


//////
//////Definizione classe
//////
class ParallelKMeans{
private:
    int K, iters, dimensions{}, total_points{};
    std::vector<Cluster> clusters;
    std::string input_dir;
    std::string output_dir;
    Point all_points;


    int getNearestClusterId(int pointIdx);
    int getNearestClusterId_Parallel(Point point);

public:
    ParallelKMeans(int K, int iterations, std::string output_dir, std::string input_dir);
    void run();
    void run_parallel(std::vector<Point> &all_points);
    void run_parallel2(std::vector<Point> &all_points);
    ~ParallelKMeans() {
        // Releases all the remaining resources allocated on the GPU
        hipDeviceReset();
    }
};
/////
/////Implementazione costruttore
/////
ParallelKMeans::ParallelKMeans(int K, int iterations, std::string output_dir, std::string input_dir):all_points(input_dir) {
    this->K = K;
    this->iters = iterations;
    this->output_dir = output_dir;
    //this->all_points = Point(input_dir);
}
void ParallelKMeans::run() {
    total_points = all_points.getDimensions();  //num totale di punti
    float centroids_shifts_sum = 0;
    float max_tollerance = 0.0001;
    //alloco memoria Host

    float *h_xval = (float *) malloc(N * sizeof(float));
    float *h_yval = (float *) malloc(N * sizeof(float));
    int *h_clusterVal = (int *) malloc(N * sizeof(int));
    //float*h_centroids =
    float *h_centroidX = (float *) malloc(K * sizeof(float));
    float *h_centroidY = (float *) malloc(K * sizeof(float));
    float *h_prevCentroidX = (float *) malloc(K * sizeof(float));
    float *h_prevCentroidY = (float *) malloc(K * sizeof(float));

    int *h_clusterSize = (int *) malloc(K * sizeof(int));
    float *h_clusterSumX = (float *) malloc(K * sizeof(float));
    float *h_clusterSumY = (float *) malloc(K * sizeof(float));
    bool *h_done = (bool *)malloc(sizeof(bool));

    //alloco memoria Device
    float *d_xval;
    float *d_yval;
    //float*h_centroids =
    int *d_clusterVal;
    float *d_centroidX;
    float *d_centroidY;
    int *d_clusterSize;
    float *d_clusterSumX;
    float *d_clusterSumY;
    bool *d_done;
    hipMalloc(&d_xval, total_points * sizeof(float));
    hipMalloc(&d_yval, total_points * sizeof(float));
    hipMalloc(&d_centroidX, K * sizeof(float));
    hipMalloc(&d_centroidY, K * sizeof(float));
    hipMalloc(&d_clusterVal, N * sizeof(int));
    hipMalloc(&d_clusterSize, K * sizeof(int));
    hipMalloc(&d_clusterSumX, K * sizeof(float));
    hipMalloc(&d_clusterSumY, K * sizeof(float));
    hipMalloc(&d_done, sizeof(bool));

    readPoints(h_xval, h_yval, h_clusterVal);       //leggo i punti dataset

    dimensions = 2;   //dimensione punto

    //std::cout<<"dimensione all_points: "<<total_points<<" dimensione clusters: "<<all_points.getDimClusters()<<" punto 1: "<<all_points.getYval(1)<< std::endl;
    // Inizializzo Clusters
    std::vector<int> used_pointIds;  //mem punti gia usati per init cluster

    bool exit = false;
    float x, y;

    used_pointIds = indexGenerator(K, N);
    for (int i = 0; i < K; i++) {

        h_centroidX[i] = h_xval[used_pointIds[i]];
        h_centroidY[i] = h_yval[used_pointIds[i]];
        //x = all_points.getXval(used_pointIds[i]);
        //y = all_points.getYval(used_pointIds[i]);
        h_clusterVal[used_pointIds[i]] = i;
        h_clusterSize[i] = 0;
    }
    std::cout << "Clusters Inizializzati = " << std::endl
              << std::endl;

    std::cout << "Eseguo Clustering K-Means..." << std::endl;

    int iter = 1;
    *h_done = true;
    //copia host -> device

    hipMemcpy(d_xval, h_xval, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_yval, h_yval, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroidX, h_centroidX, K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroidY, h_centroidY, K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterSize, h_clusterSize, K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterSumX, h_centroidX, K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterSumY, h_centroidY, K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_done, h_done,sizeof(bool), hipMemcpyHostToDevice);

    while (true) {

        std::cout << "Iter - " << iter << "/" << iters << std::endl;
        bool done = true;
        clusterAssignment<<<(N + TPB - 1) / TPB, TPB>>>(d_xval, d_yval, d_clusterVal, d_centroidX, d_centroidY, K, d_done);

        //setto a 0 la sommma dei punti appartenenti ai cluster, per iniziare aggiornamento centroide
        hipMemset(d_clusterSumX, 0.0, K * sizeof(float));
        hipMemset(d_clusterSumY, 0.0, K * sizeof(float));

        clusterPointsSum<<<(N + TPB - 1) / TPB, TPB>>>(d_xval, d_yval, d_clusterVal, d_clusterSumX, d_clusterSumY,
                                                       d_clusterSize);

        //copia variabili aggiornate da device -> host

        hipMemcpy(h_clusterSumX, d_clusterSumX, K * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_clusterSumY, d_clusterSumY, K * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_clusterSize, d_clusterSize, K * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_done, d_done, sizeof(bool), hipMemcpyDeviceToHost);

        hipMemset(d_clusterSize, 0, K * sizeof(int));

        //Ricalcolo nuovi centroidi per ogni cluster
        for (int i = 0; i < K; i++) {
            h_centroidX[i] = h_clusterSumX[i] / h_clusterSize[i];
            h_centroidY[i] = h_clusterSumY[i] / h_clusterSize[i];

        }

        hipMemcpy(d_centroidX, h_centroidX, K * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_centroidY, h_centroidY, K * sizeof(float), hipMemcpyHostToDevice);




        float deltaX, deltaY = 0;
        for(int i= 0; i < K; i++)
        {
                deltaX = h_centroidX[i] - h_prevCentroidX[i];
                deltaY = h_centroidY[i] - h_prevCentroidY[i];
                double percentageShiftX = std::abs((deltaX / h_centroidX[i]) * 100.0);
                double percentageShiftY = std::abs((deltaY / h_centroidY[i]) * 100.0);
                std::cout<< "perX: "<< percentageShiftX << " percY: "<< percentageShiftY<< std::endl;
                if(percentageShiftX > max_tollerance || percentageShiftY > max_tollerance)
                {
                    done = false;

                }
            }

        for (int i = 0; i < K; i++) {
            h_prevCentroidX[i] = h_centroidX[i];
            h_prevCentroidY[i] = h_centroidY[i];

        }
        if (done || iter >= iters) {
            std::cout << "Clustering completed in iteration : " << iter << std::endl
                      << std::endl;
            break;
        }
        iter++;

    }
    hipMemcpy(h_clusterVal, d_clusterVal, N * sizeof(int), hipMemcpyDeviceToHost);


    ///scrivo i risultati clustering su file
    //scrittura centroidi
    std::ofstream outfile;
    //std::cout << std::filesystem::current_path().string() << std::endl;
    //std::cout << output_dir + "/" + "clusters.txt" << std::endl;
    outfile.open(output_dir + "/" + "clusters.txt");
    if (outfile.is_open()) {
        for (int i = 0; i < K; i++) {
            //std::cout <<  i << " cluster contiene: "<< clusters[i].getSize() <<std::endl;
            std::cout << "Cluster " << i << " centroid : ";

            std::cout << h_centroidX[i] << " "<< h_centroidY[i] << std::endl;    // Output console

            outfile << h_centroidX[i] << " " << h_centroidY[i]; // Output file
            outfile << std::endl;
        }
        std::cout << std::endl;
        outfile << std::endl;
        outfile.close();
    } else {
        std::cout << "Error: Unable to write to clusters.txt";
    }
    //scrittura punti e cluster di appartenenza
    outfile.open(output_dir + "/" + "clustering.txt");
    for (int i = 0; i < N; i++) {
        // indice cluster è ID-1
        if (outfile.is_open()) {

            outfile << h_xval[i] << " " <<  h_yval[i] << " "
                    << h_clusterVal[i]  ;// Output to file
        }

        outfile << std::endl;

    }
    outfile.close();

    hipFree(d_xval);
    hipFree(d_yval);
    hipFree(d_clusterVal);
    hipFree(d_centroidX);
    hipFree(d_centroidY);
    hipFree(d_clusterSize);

    free(h_xval);
    free(h_yval);
    free(h_clusterVal);
    free(h_centroidX);
    free(h_centroidY);
    free(h_clusterSize);

}



float averageParallelExecutions(int K, int iters, std::string output_dir, std::string input_dir)
{
    float mediaS, mediaP;
    float sum;

    for(int i  = 0; i < 2; i++ ) {
        auto start = std::chrono::high_resolution_clock::now();
        ParallelKMeans kmeans(K, iters, output_dir, input_dir);
        //kmeans.run_parallel2(all_points);
        kmeans.run();
        //float end = omp_get_wtime( );
        auto end = std::chrono::high_resolution_clock::now();

        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << "Tempo di esecuzione parallela: " << duration.count() << " millisecondi" << std::endl;
        sum += static_cast<float>(duration.count());
        std::cout << "<<------------------------------>>" << std::endl;

    }
    mediaP = static_cast<float>(sum)/2;
    return mediaP;
}

float averageSeqExecutions(int K, int iters, std::string output_dir, std::string input_dir)
{
    float mediaS;
    float sum;

    for(int i  = 0; i < 2; i++ ) {
        auto start = std::chrono::high_resolution_clock::now();
        SequentialKMeans kmeans(K, iters, output_dir, input_dir);
        kmeans.run();
        //float end = omp_get_wtime( );
        auto end = std::chrono::high_resolution_clock::now();

        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << "Tempo di esecuzione Sequenziale: " << duration.count() << " millisecondi" << std::endl;
        sum += static_cast<float>(duration.count());
        std::cout << "<<------------------------------>>" << std::endl;

    }
    mediaS = static_cast<float>(sum)/2;
    return mediaS;
}

int main() {
    std::string output_dir = "../cmake-build-debug/cluster_details";   //dir output
    int K = 3;                               //numero cluster
    std::string input_dir= "input1.txt";

    // Avvio il clustering
    int iters = 100;


    auto mediaS = averageSeqExecutions(K, iters, output_dir, input_dir);
    auto mediaP = averageParallelExecutions(K, iters, output_dir, input_dir);
    std::cout << "Media esecuzione Sequenziale : " << mediaS << std::endl;
    std::cout << "Media esecuzione Parallela : " << mediaP << std::endl;

    float speedup = static_cast<float>(mediaS) / static_cast<float>(mediaP);
    std::cout << "Speedup: " << speedup << std::endl;

    return 0;
}
