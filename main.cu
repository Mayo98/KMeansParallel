#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include "Cluster.h"
#include "Point.h"
#include <cmath>
#include <vector>
#include <string>
#include <fstream>
#include <chrono>
#include <filesystem>
#include "SequentialKMeans.h"
#include <algorithm>


#define N 400000
#define TPB 128
static std::vector<int> used_pointIds;
using namespace std;

__host__ void readPoints(double *h_xval, double *h_yval, int* h_clusters) {

    std::string tmp = "";
    string line;
    std::cout << std::filesystem::current_path().string() << "Ciao" << std::endl;
    ifstream infile("../cmake-build-debug/input1.txt");
    if (!infile.is_open()) {
        cout << "Error: Failed to open file." << endl;
        return;
    }
    int j = 0;
    while (getline(infile, line)) {
        for (int i = 0; i < static_cast<int>(line.length()); i++) {
            if ((48 <= static_cast<int>(line[i]) && static_cast<int>(line[i]) <= 57) || line[i] == '.' ||
                line[i] == '+' || line[i] == '-' || line[i] == 'e') {
                tmp += line[i];

            } else if (!tmp.empty()) {


                h_xval[i] = std::stod(tmp);
                //xval.push_back(std::stod(tmp));
                tmp = "";
            }
        }
        if (!tmp.empty()) {

           h_yval[j] = std::stod(tmp);
            //yval.push_back(std::stod(tmp));
            tmp = "";
        }
       h_clusters[j] = -1; //inizializzo anche il vettore clusters a -1
       j++;
    }
    //std::cout << "dim clusters: " << h_clusters << std::endl;

    cout << "\nDataset fetched!" << endl
         << endl;
    //std::cout << "Punti totali letti : " << xval.size() << std::endl;
}



static bool first = true;
std::vector<int>indexGenerator(int K, int total_points){
    //mem punti gia usati per init cluster
    if (first) {
        for (int i = 0; i < K; i++) {
            while (true) {
                int index = rand() % total_points;

                //check se index gia usato per un altro cluster

                if (std::find(used_pointIds.begin(), used_pointIds.end(), index) ==
                    used_pointIds.end()) {
                    used_pointIds.push_back(index);
                    //creo un cluster con avete centroide il punto attuale
                    break;
                }
            }
        }
        first = false;
    }
    return used_pointIds;
}
__global__ void clusterAssignment(const double *d_xval, const double *d_yval, int *d_clusterval, const double *d_centroidX, const double *d_centroidY, int K){
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= N) return;
    float min_dist = INFINITY;
    int closest_centroid = 0;

    double dist;
    for(int i = 0; i < K; i++)
    {
        double sum = 0.0;
        sum += pow(d_centroidX[i] - d_xval[idx], 2.0);
        sum += pow(d_centroidY[i] - d_yval[idx], 2.0);
        dist = sqrt(sum);
        if(dist < min_dist){
            min_dist = dist;
            closest_centroid = i;
        }
    }
    //assegno id-cluster al thread corrente
    d_clusterval[idx] = closest_centroid;
}

__global__ void clusterPointsSum(double* d_xval, double* d_yval, int* d_clusterVal, double* d_clusterSumX, double* d_clusterSumY, int* d_clusterSize){
    //indice del thread a livello grid

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    int clusterId = d_clusterVal[idx];
    //sommo tutti i punti appartenenti ai clusters
    atomicAdd(&(d_clusterSumX[clusterId]), d_xval[idx]);
    atomicAdd(&(d_clusterSumY[clusterId]), d_yval[idx]);
    atomicAdd(&(d_clusterSize[clusterId]), 1);

}


//////
//////Definizione classe
//////
class ParallelKMeans{
private:
    int K, iters, dimensions{}, total_points{};
    std::vector<Cluster> clusters;
    std::string input_dir;
    std::string output_dir;
    Point all_points;


    int getNearestClusterId(int pointIdx);
    int getNearestClusterId_Parallel(Point point);

public:
    ParallelKMeans(int K, int iterations, std::string output_dir, std::string input_dir);
    void run();
    void run_parallel(std::vector<Point> &all_points);
    void run_parallel2(std::vector<Point> &all_points);
    ~ParallelKMeans() {
        // Releases all the remaining resources allocated on the GPU
        hipDeviceReset();
    }
};
/////
/////Implementazione costruttore
/////
ParallelKMeans::ParallelKMeans(int K, int iterations, std::string output_dir, std::string input_dir):all_points(input_dir) {
    this->K = K;
    this->iters = iterations;
    this->output_dir = output_dir;
    //this->all_points = Point(input_dir);
}
void ParallelKMeans::run() {
    total_points = all_points.getDimensions();  //num totale di punti

    //alloco memoria Host

    double *h_xval = (double *) malloc(N * sizeof(double));
    double *h_yval = (double *) malloc(N * sizeof(double));
    int *h_clusterVal = (int *) malloc(N * sizeof(int));
    //double*h_centroids =
    double *h_centroidX = (double *) malloc(K * sizeof(double));
    double *h_centroidY = (double *) malloc(K * sizeof(double));
    int *h_clusterSize = (int *) malloc(K * sizeof(int));
    double *h_clusterSumX = (double *) malloc(K * sizeof(double));
    double *h_clusterSumY = (double *) malloc(K * sizeof(double));

    //alloco memoria Device
    double *d_xval;
    double *d_yval;
    //double*h_centroids =
    int *d_clusterVal;
    double *d_centroidX;
    double *d_centroidY;
    int *d_clusterSize;
    double *d_clusterSumX;
    double *d_clusterSumY;

    hipMalloc(&d_xval, total_points * sizeof(double));
    hipMalloc(&d_yval, total_points * sizeof(double));
    hipMalloc(&d_centroidX, K * sizeof(double));
    hipMalloc(&d_centroidY, K * sizeof(double));
    hipMalloc(&d_clusterVal, N * sizeof(int));
    hipMalloc(&d_clusterSize, K * sizeof(int));
    hipMalloc(&d_clusterSumX, K * sizeof(double));
    hipMalloc(&d_clusterSumY, K * sizeof(double));

    readPoints(h_xval, h_yval, h_clusterVal);       //leggo i punti dataset

    dimensions = 2;   //dimensione punto

    //std::cout<<"dimensione all_points: "<<total_points<<" dimensione clusters: "<<all_points.getDimClusters()<<" punto 1: "<<all_points.getYval(1)<< std::endl;
    // Inizializzo Clusters
    std::vector<int> used_pointIds;  //mem punti gia usati per init cluster

    bool exit = false;
    double x, y;

    used_pointIds = indexGenerator(K, N);
    for (int i = 0; i < K; i++) {

        h_centroidX[i] = h_xval[used_pointIds[i]];
        h_centroidY[i] = h_yval[used_pointIds[i]];
        //x = all_points.getXval(used_pointIds[i]);
        //y = all_points.getYval(used_pointIds[i]);
        h_clusterVal[used_pointIds[i]] = i;
        h_clusterSize[i] = 0;
        //all_points.setCluster(used_pointIds[i], i);
        //Cluster cluster(i,used_pointIds[i], x, y);  //creo un cluster avente centroide il punto attuale
        //clusters.push_back(cluster);
    }
    std::cout << "Clusters Inizializzati = " << std::endl
              << std::endl;

    std::cout << "Eseguo Clustering K-Means..." << std::endl;

    int iter = 1;

    //copia host -> device

    hipMemcpy(d_xval, h_xval, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_yval, h_yval, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroidX, h_centroidX, K * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroidY, h_centroidY, K * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterSize, h_clusterSize, K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterSumX, h_centroidX, K * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterSumY, h_centroidY, K * sizeof(double), hipMemcpyHostToDevice);


    while (iter <= iters) {
        std::cout << "Iter - " << iter << "/" << iters << std::endl;
        bool done = true;
        clusterAssignment<<<(N + TPB - 1) / TPB, TPB>>>(d_xval, d_yval, d_clusterVal, d_centroidX, d_centroidY, K);

        //setto a 0 la sommma dei punti appartenenti ai cluster, per iniziare aggiornamento centroide
        hipMemset(d_clusterSumX, 0.0, K * sizeof(double));
        hipMemset(d_clusterSumY, 0.0, K * sizeof(double));

        clusterPointsSum<<<(N + TPB - 1) / TPB, TPB>>>(d_xval, d_yval, d_clusterVal, d_clusterSumX, d_clusterSumY,
                                                       d_clusterSize);

        //copia variabili aggiornate da device -> host

        hipMemcpy(h_clusterSumX, d_clusterSumX, K * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_clusterSumY, d_clusterSumY, K * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_clusterSize, d_clusterSize, K * sizeof(int), hipMemcpyDeviceToHost);

        hipMemset(d_clusterSize, 0, K * sizeof(int));

        //Ricalcolo nuovi centroidi per ogni cluster
        for (int i = 0; i < K; i++) {
            h_centroidX[i] = h_clusterSumX[i] / h_clusterSize[i];
            h_centroidY[i] = h_clusterSumY[i] / h_clusterSize[i];
        }

        hipMemcpy(d_centroidX, h_centroidX, K * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_centroidY, h_centroidY, K * sizeof(double), hipMemcpyHostToDevice);

        iter++;

    }
    hipMemcpy(h_clusterVal, d_clusterVal, N * sizeof(int), hipMemcpyDeviceToHost);


    ///scrivo i risultati clustering su file
    //scrittura centroidi
    std::ofstream outfile;
    std::cout << std::filesystem::current_path().string() << std::endl;
    std::cout << output_dir + "/" + "clusters.txt" << std::endl;
    outfile.open(output_dir + "/" + "clusters.txt");
    if (outfile.is_open()) {
        for (int i = 0; i < K; i++) {
            //std::cout <<  i << " cluster contiene: "<< clusters[i].getSize() <<std::endl;
            std::cout << "Cluster " << i << " centroid : ";

            std::cout << h_centroidX[i] << " ";    // Output console
            std::cout << h_centroidY[i] << " ";
            outfile << h_centroidX[i] << " " << h_centroidY[i]; // Output file
        }
        std::cout << std::endl;
        outfile << std::endl;
        outfile.close();
    } else {
        std::cout << "Error: Unable to write to clusters.txt";
    }
    //scrittura punti e cluster di appartenenza
    outfile.open(output_dir + "/" + "clustering.txt");
    for (int i = 0; i < N; i++) {
        // indice cluster è ID-1
        if (outfile.is_open()) {

            outfile << h_xval[i] << " " <<  h_yval[i] << " "
                    << h_clusterVal[i]  ;// Output to file
        }

        outfile << std::endl;

    }
    outfile.close();
}



double averageParallelExecutions(int K, int iters, std::string output_dir, std::string input_dir)
{
    double mediaS, mediaP;
    double sum;

    for(int i  = 0; i < 2; i++ ) {
        auto start = std::chrono::high_resolution_clock::now();
        ParallelKMeans kmeans(K, iters, output_dir, input_dir);
        //kmeans.run_parallel2(all_points);
        kmeans.run();
        //double end = omp_get_wtime( );
        auto end = std::chrono::high_resolution_clock::now();

        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << "Tempo di esecuzione parallela: " << duration.count() << " millisecondi" << std::endl;
        sum += static_cast<double>(duration.count());
        std::cout << "<<------------------------------>>" << std::endl;

    }
    mediaP = static_cast<double>(sum)/2;
    return mediaP;
}

double averageSeqExecutions(int K, int iters, std::string output_dir, std::string input_dir)
{
    double mediaS;
    double sum;

    for(int i  = 0; i < 2; i++ ) {
        auto start = std::chrono::high_resolution_clock::now();
        SequentialKMeans kmeans(K, iters, output_dir, input_dir);
        kmeans.run();
        //double end = omp_get_wtime( );
        auto end = std::chrono::high_resolution_clock::now();

        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        std::cout << "Tempo di esecuzione Sequenziale: " << duration.count() << " millisecondi" << std::endl;
        sum += static_cast<double>(duration.count());
        std::cout << "<<------------------------------>>" << std::endl;

    }
    mediaS = static_cast<double>(sum)/2;
    return mediaS;
}

int main() {
    std::string output_dir = "../cmake-build-debug/cluster_details";   //dir output
    int K = 4;                               //numero cluster
    std::string input_dir= "input1.txt";

    // Avvio il clustering
    int iters = 100;


    //auto mediaS = averageSeqExecutions(K, iters, output_dir, input_dir);
    auto mediaP = averageParallelExecutions(K, iters, output_dir, input_dir);
    //std::cout << "Media esecuzione Sequenziale : " << mediaS << std::endl;
    std::cout << "Media esecuzione Parallela : " << mediaP << std::endl;

    //double speedup = static_cast<double>(mediaS) / static_cast<double>(mediaP);
    //std::cout << "Speedup: " << speedup << std::endl;
    return 0;
}
